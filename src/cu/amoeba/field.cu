#include "hip/hip_runtime.h"
#include "ff/modamoeba.h"
#include "ff/cumodamoeba.h"
#include "ff/image.h"
#include "ff/pme.h"
#include "ff/spatial.h"
#include "ff/switch.h"
#include "seq/launch.h"
#include "seq/pair_field.h"
#include "seq/triangle.h"

namespace tinker {
__global__
void dfieldEwaldRecipSelfP2_cu1(int n, real (*restrict field)[3], real term, const real (*restrict rpole)[MPL_TOTAL],
   const real (*restrict cphi)[10])
{
   for (int i = ITHREAD; i < n; i += STRIDE) {
      real dix = rpole[i][MPL_PME_X];
      real diy = rpole[i][MPL_PME_Y];
      real diz = rpole[i][MPL_PME_Z];
      field[i][0] += (-cphi[i][1] + term * dix);
      field[i][1] += (-cphi[i][2] + term * diy);
      field[i][2] += (-cphi[i][3] + term * diz);
   }
}

void dfieldEwaldRecipSelfP2_cu(real (*field)[3])
{
   const PMEUnit pu = ppme_unit;
   const real aewald = pu->aewald;
   const real term = aewald * aewald * aewald * 4 / 3 / sqrtpi;

   launch_k1s(g::s0, n, dfieldEwaldRecipSelfP2_cu1, n, field, term, rpole, cphi);
}

#include "dfield_cu1.cc"

void dfieldEwaldReal_cu(real (*field)[3], real (*fieldp)[3])
{
   const auto& st = *mspatial_v2_unit;
   const real off = switchOff(Switch::EWALD);

   const PMEUnit pu = ppme_unit;
   const real aewald = pu->aewald;

   int ngrid = gpuGridSize(BLOCK_DIM);
   ngrid *= BLOCK_DIM;
   int nparallel = std::max(st.niak, st.nakpl) * WARP_SIZE;
   nparallel = std::max(nparallel, ngrid);
   launch_k1s(g::s0, nparallel, dfield_cu1<EWALD>, //
      st.n, TINKER_IMAGE_ARGS, off, st.si3.bit0, ndpexclude, dpexclude, dpexclude_scale, st.x, st.y, st.z, st.sorted,
      st.nakpl, st.iakpl, st.niak, st.iak, st.lst, field, fieldp, aewald);
}

void dfieldNonEwald_cu(real (*field)[3], real (*fieldp)[3])
{
   const auto& st = *mspatial_v2_unit;
   const real off = switchOff(Switch::MPOLE);

   darray::zero(g::q0, n, field, fieldp);
   int ngrid = gpuGridSize(BLOCK_DIM);
   ngrid *= BLOCK_DIM;
   int nparallel = std::max(st.niak, st.nakpl) * WARP_SIZE;
   nparallel = std::max(nparallel, ngrid);
   launch_k1s(g::s0, nparallel, dfield_cu1<NON_EWALD>, //
      st.n, TINKER_IMAGE_ARGS, off, st.si3.bit0, ndpexclude, dpexclude, dpexclude_scale, st.x, st.y, st.z, st.sorted,
      st.nakpl, st.iakpl, st.niak, st.iak, st.lst, field, fieldp, 0);
}
}

namespace tinker {
__global__
void ufieldEwaldRecipSelfP1_cu1(int n, const real (*restrict uind)[3], const real (*restrict uinp)[3],
   real (*restrict field)[3], real (*restrict fieldp)[3], const real (*restrict fdip_phi1)[10],
   const real (*restrict fdip_phi2)[10], real term, int nfft1, int nfft2, int nfft3, TINKER_IMAGE_PARAMS)
{
   real a[3][3];
   a[0][0] = nfft1 * recipa.x;
   a[1][0] = nfft2 * recipb.x;
   a[2][0] = nfft3 * recipc.x;
   a[0][1] = nfft1 * recipa.y;
   a[1][1] = nfft2 * recipb.y;
   a[2][1] = nfft3 * recipc.y;
   a[0][2] = nfft1 * recipa.z;
   a[1][2] = nfft2 * recipb.z;
   a[2][2] = nfft3 * recipc.z;

   if (uinp) {
      for (int i = ITHREAD; i < n; i += STRIDE) {
         for (int j = 0; j < 3; ++j) {
            real df1 = a[0][j] * fdip_phi1[i][1] + a[1][j] * fdip_phi1[i][2] + a[2][j] * fdip_phi1[i][3];
            real df2 = a[0][j] * fdip_phi2[i][1] + a[1][j] * fdip_phi2[i][2] + a[2][j] * fdip_phi2[i][3];
            field[i][j] += (term * uind[i][j] - df1);
            fieldp[i][j] += (term * uinp[i][j] - df2);
         }
      }
   } else {
      for (int i = ITHREAD; i < n; i += STRIDE) {
         for (int j = 0; j < 3; ++j) {
            real df1 = a[0][j] * fdip_phi1[i][1] + a[1][j] * fdip_phi1[i][2] + a[2][j] * fdip_phi1[i][3];
            field[i][j] += (term * uind[i][j] - df1);
         }
      }
   }
}

void ufieldEwaldRecipSelfP1_cu(const real (*uind)[3], const real (*uinp)[3], //
   real (*field)[3], real (*fieldp)[3])
{
   const PMEUnit pu = ppme_unit;
   const real aewald = pu->aewald;
   const real term = aewald * aewald * aewald * 4 / 3 / sqrtpi;
   const int nfft1 = pu->nfft1;
   const int nfft2 = pu->nfft2;
   const int nfft3 = pu->nfft3;

   launch_k1s(g::s0, n, ufieldEwaldRecipSelfP1_cu1, n, uind, uinp, field, fieldp, fdip_phi1, fdip_phi2, term, nfft1,
      nfft2, nfft3, TINKER_IMAGE_ARGS);
}

#include "ufield_cu1.cc"

void ufieldEwaldReal_cu(const real (*uind)[3], const real (*uinp)[3], real (*field)[3], real (*fieldp)[3])
{
   const auto& st = *mspatial_v2_unit;
   const real off = switchOff(Switch::EWALD);

   const PMEUnit pu = ppme_unit;
   const real aewald = pu->aewald;

   int ngrid = gpuGridSize(BLOCK_DIM);
   ngrid *= BLOCK_DIM;
   int nparallel = std::max(st.niak, st.nakpl) * WARP_SIZE;
   nparallel = std::max(nparallel, ngrid);
   launch_k1s(g::s0, nparallel, ufield_cu1<EWALD>, //
      st.n, TINKER_IMAGE_ARGS, off, st.si4.bit0, nuexclude, uexclude, uexclude_scale, st.x, st.y, st.z, st.sorted,
      st.nakpl, st.iakpl, st.niak, st.iak, st.lst, uind, uinp, field, fieldp, aewald);
}

void ufieldNonEwald_cu(const real (*uind)[3], const real (*uinp)[3], real (*field)[3], real (*fieldp)[3])
{
   const auto& st = *mspatial_v2_unit;
   const real off = switchOff(Switch::MPOLE);

   darray::zero(g::q0, n, field, fieldp);
   int ngrid = gpuGridSize(BLOCK_DIM);
   ngrid *= BLOCK_DIM;
   int nparallel = std::max(st.niak, st.nakpl) * WARP_SIZE;
   nparallel = std::max(nparallel, ngrid);
   launch_k1s(g::s0, nparallel, ufield_cu1<NON_EWALD>, //
      st.n, TINKER_IMAGE_ARGS, off, st.si4.bit0, nuexclude, uexclude, uexclude_scale, st.x, st.y, st.z, st.sorted,
      st.nakpl, st.iakpl, st.niak, st.iak, st.lst, uind, uinp, field, fieldp, 0);
}
}